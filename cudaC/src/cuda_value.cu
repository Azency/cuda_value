#include "hip/hip_runtime.h"
#include "cuda_value.h"
// #include "config.h"

float h_MIN_X, h_MIN_Y, h_MIN_Z, h_MIN_W;
float h_MAX_X, h_MAX_Y, h_MAX_Z, h_MAX_W;
int h_SIZE_X, h_SIZE_Y, h_SIZE_Z, h_SIZE_E, h_SIZE_W;
int h_sWEYZX, h_sEYZX, h_sYZX, h_sZX, h_sX;

float h_SCALE_TO_INT_X, h_SCALE_TO_INT_Y, h_SCALE_TO_INT_Z;

__constant__ float d_MIN_X, d_MIN_Y, d_MIN_Z, d_MIN_W;
__constant__ float d_MAX_X, d_MAX_Y, d_MAX_Z, d_MAX_W;
__constant__ int d_SIZE_X, d_SIZE_Y, d_SIZE_Z, d_SIZE_E, d_SIZE_W;
__constant__ int d_sWEYZX, d_sEYZX, d_sYZX, d_sZX, d_sX;

__constant__ float d_SCALE_TO_INT_X, d_SCALE_TO_INT_Y, d_SCALE_TO_INT_Z;


float h_A1,h_P, h_INITIAL_INVESTMENT, h_DELTA_T;
__constant__ float d_A1, d_A2, d_R, d_MU, d_SIGMA, d_P, d_INITIAL_INVESTMENT, d_DELTA_T;
__constant__ int d_MOTECALO_NUMS;

float *d_X, *d_Y, *d_Z, *d_W, *d_V, *d_V_tp1, *d_results;
int *d_E;

__constant__ float *d_d_X, *d_d_Y, *d_d_Z, *d_d_W, *d_d_V, *d_d_V_tp1, *d_d_results;
__constant__ int *d_d_E;

// 2. === 使用 hipArray 分配设备内存 ===
hipArray_t cuArray0;
hipArray_t cuArray1;
// 描述我们数据的格式（单通道32位浮点数）
hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

hipTextureObject_t texObj0 = 0;
hipTextureObject_t texObj1 = 0;
hipSurfaceObject_t surfObj0 = 0;
hipSurfaceObject_t surfObj1 = 0;

__constant__ hipTextureObject_t d_texObj0 = 0;
__constant__ hipTextureObject_t d_texObj1 = 0;
__constant__ hipSurfaceObject_t d_surfObj0 = 0;
__constant__ hipSurfaceObject_t d_surfObj1 = 0;

// 随机数生成器
hiprandStatePhilox4_32_10_t* d_rng_states;




// 生成随机数
__global__ void setup(hiprandStatePhilox4_32_10_t *state, unsigned long seed, int PATHS)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= PATHS) return;
    /* sequence=tid, offset=0 → 线程独立子流 */
    hiprand_init(seed, tid, 0, &state[tid]);
}


__device__ int IDX_V(int e, int y, int z, int x) {
    // if(x*sYZEW + y*sZEW + z*sEW + e*sW + e >= SIZE_X * SIZE_Y * SIZE_Z * SIZE_E) {
    //     printf("Error: Index out of bounds\n");
    //     exit(1);
    // }
    int res = e*d_sYZX + y*d_sZX + z*d_sX + x;
    return res;
}

__host__ int h_IDX_V(int e, int y, int z, int x){
    int res = e*h_sYZX + y*h_sZX + z*h_sX + x;
    return res;
}


void init_global_config(
    float min_X, float max_X, int size_X,
    float min_Y, float max_Y, int size_Y,
    float min_Z, float max_Z, int size_Z,
    int min_E, int max_E, int size_E,
    float min_W, float max_W, int size_W,
    float a1, float a2, float r, float mu, float sigma, int motecalo_nums, float p, float initial_investment
){
    
    h_A1 = a1;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_A1), &h_A1, sizeof(float));
    float h_A2 = a2;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_A2), &h_A2, sizeof(float));
    float h_R = r;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_R), &h_R, sizeof(float));
    float h_MU = mu;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MU), &h_MU, sizeof(float));
    float h_SIGMA = sigma;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIGMA), &h_SIGMA, sizeof(float));
    int h_MOTECALO_NUMS = motecalo_nums;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MOTECALO_NUMS), &h_MOTECALO_NUMS, sizeof(int));
    h_P = p;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_P), &h_P, sizeof(float));
    h_INITIAL_INVESTMENT = initial_investment;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_INITIAL_INVESTMENT), &h_INITIAL_INVESTMENT, sizeof(float));
    h_DELTA_T = 1.0f/h_P;
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_DELTA_T), &h_DELTA_T, sizeof(float));
    
    h_MIN_X = min_X;
    h_MAX_X = max_X;
    h_SIZE_X = size_X;
    h_MIN_Y = min_Y;
    h_MAX_Y = max_Y;
    h_SIZE_Y = size_Y;
    h_MIN_Z = min_Z;
    h_MAX_Z = max_Z;
    h_SIZE_Z = size_Z;
    h_SIZE_E = size_E;
    h_MIN_W = min_W;
    h_MAX_W = max_W;
    h_SIZE_W = size_W;

    h_sWEYZX = size_W * size_E * size_Y * size_Z * size_X;
    h_sEYZX = size_E * size_Y * size_Z * size_X;
    h_sYZX = size_Y * size_Z * size_X;
    h_sZX = size_Z * size_X;
    h_sX = size_X;


    h_SCALE_TO_INT_X = (float)(size_X-1) / (max_X - min_X);
    h_SCALE_TO_INT_Y = (float)(size_Y-1) / (max_Y - min_Y);
    h_SCALE_TO_INT_Z = (float)(size_Z-1) / (max_Z - min_Z);

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_X), &h_MIN_X, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_X), &h_MAX_X, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_Y), &h_MIN_Y, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_Y), &h_MAX_Y, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_Z), &h_MIN_Z, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_Z), &h_MAX_Z, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_W), &h_MIN_W, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_W), &h_MAX_W, sizeof(float));
    
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_X), &h_SIZE_X, sizeof(int)); 
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_Y), &h_SIZE_Y, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_Z), &h_SIZE_Z, sizeof(int));   
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_E), &h_SIZE_E, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_W), &h_SIZE_W, sizeof(int));


    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sWEYZX), &h_sWEYZX, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sEYZX), &h_sEYZX, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sYZX), &h_sYZX, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sZX), &h_sZX, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sX), &h_sX, sizeof(int));

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SCALE_TO_INT_X), &h_SCALE_TO_INT_X, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SCALE_TO_INT_Y), &h_SCALE_TO_INT_Y, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SCALE_TO_INT_Z), &h_SCALE_TO_INT_Z, sizeof(float));


 

}

void init_global_XYZEW_V() {
   // 初始化XYZEW_V
    float *h_X = (float *)malloc(h_SIZE_X * sizeof(float));
    float *h_Y = (float *)malloc(h_SIZE_Y * sizeof(float));
    float *h_Z = (float *)malloc(h_SIZE_Z * sizeof(float));
    int   *h_E = (int   *)malloc(h_SIZE_E * sizeof(int));
    float *h_W = (float *)malloc(h_SIZE_W * sizeof(float));
    float *h_V = (float *)malloc(h_sEYZX * sizeof(float));
    
    for (int i = 0; i < h_SIZE_X; i++) {
        h_X[i] = h_MIN_X + float(h_MAX_X - h_MIN_X) * i / (h_SIZE_X - 1);
    }
    for (int i = 0; i < h_SIZE_Y; i++) {
        h_Y[i] = h_MIN_Y + float(h_MAX_Y - h_MIN_Y) * i / (h_SIZE_Y - 1);
    }
    for (int i = 0; i < h_SIZE_Z; i++) {
        h_Z[i] = h_MIN_Z + float(h_MAX_Z - h_MIN_Z) * i / (h_SIZE_Z - 1);
    }
    for (int i = 0; i < h_SIZE_E; i++) {
        h_E[i] = i;
    }
    for (int i = 0; i < h_SIZE_W; i++) {
        h_W[i] = h_MIN_W + float(h_MAX_W - h_MIN_W) * i / (h_SIZE_W - 1);
    }

    // 初始化 V 数组
    for (int x = 0; x < h_SIZE_X; x++) {
        for (int y = 0; y < h_SIZE_Y; y++) {
            for (int z = 0; z < h_SIZE_Z; z++) {
                float min_ZY = fminf(h_Z[z], h_Y[y]);
                float term = (h_Y[y] <= min_ZY) ? 
                            h_Y[y] : 
                            h_Y[y] - h_A1 * (h_Y[y] - min_ZY);
                float result = fmaxf(h_X[x], term);
                
                // 对 E 的两个维度都赋值
                h_V[h_IDX_V(0, y, z, x)] = result;
                h_V[h_IDX_V(1, y, z, x)] = result;

            }
        }
    }

    printf("V array initialized\n");

    // 分配设备内存
    hipMalloc(&d_X, h_SIZE_X * sizeof(float));
    hipMalloc(&d_Y, h_SIZE_Y * sizeof(float));
    hipMalloc(&d_Z, h_SIZE_Z * sizeof(float)); 
    hipMalloc(&d_E, h_SIZE_E * sizeof(int));
    hipMalloc(&d_W, h_SIZE_W * sizeof(float));
    hipMalloc(&d_V, h_sEYZX * sizeof(float));
    hipMalloc(&d_V_tp1, h_sEYZX * sizeof(float));
    hipMalloc(&d_results, h_sWEYZX * sizeof(float));
    printf("cudamalloc done\n");
    // 检查内存分配是否成功
    // if (!d_X || !d_Y || !d_Z || !d_E || !d_W || !d_V || !d_V_tp1 || !d_results) {
    //     printf("Error: Failed to allocate device memory\n");
    //     clean_global_XYZEW_V();
    //     exit(1);
    // }

    // 复制数据到设备
    hipMemcpy(d_X, h_X, h_SIZE_X * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, h_SIZE_Y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, h_Z, h_SIZE_Z * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_E, h_E, h_SIZE_E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_W, h_W, h_SIZE_W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, h_sEYZX * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V_tp1, h_V, h_sEYZX * sizeof(float), hipMemcpyHostToDevice);
    

    // 将主机端指针值复制到设备端全局变量
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_X), &d_X, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_Y), &d_Y, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_Z), &d_Z, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_E), &d_E, sizeof(int*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_W), &d_W, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_V), &d_V, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_V_tp1), &d_V_tp1, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_results), &d_results, sizeof(float*));

    printf("init_global_XYZEW_V is done\n");

    init_texture_surface_object();
    printf("init_texture_surface_object is done\n");

    // 释放主机内存
    free(h_X);
    free(h_Y);
    free(h_Z);
    free(h_E);
    free(h_W);
    free(h_V);

    // 设置随机数生成器
    hipMalloc(&d_rng_states,  h_sWEYZX*sizeof(*d_rng_states));
}

// 清理函数
void clean_global_XYZEW_V() {
    if (d_X) hipFree(d_X);
    if (d_Y) hipFree(d_Y);
    if (d_Z) hipFree(d_Z);
    if (d_E) hipFree(d_E);
    if (d_W) hipFree(d_W);
    if (d_V) hipFree(d_V);
    if (d_V_tp1) hipFree(d_V_tp1);
    if (d_results) hipFree(d_results);
    if (cuArray0) hipFreeArray(cuArray0);
    if (cuArray1) hipFreeArray(cuArray1);
    if (d_rng_states) hipFree(d_rng_states);

    d_X = nullptr;
    d_Y = nullptr;
    d_Z = nullptr;
    d_E = nullptr;
    d_W = nullptr;
    d_V = nullptr;
    d_V_tp1 = nullptr;
    d_results = nullptr;

    printf("clean_global_XYZEW_V is done\n");
}   

void init_random_state() {
    
    setup<<<(h_sWEYZX+1023)/1024,1024>>>(d_rng_states, 101, h_sWEYZX);
}

void init_texture_surface_object() {
    
    hipExtent extent = make_hipExtent(h_SIZE_X, h_SIZE_Z, h_SIZE_Y);
    hipMalloc3DArray(&cuArray0, &channelDesc, extent, hipArrayDefault);
    hipMalloc3DArray(&cuArray1, &channelDesc, extent, hipArrayDefault);

    hipMemcpy3DParms copyParams = {0};
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;

    copyParams.dstArray = cuArray0;
    copyParams.srcPtr = make_hipPitchedPtr(d_V, (h_SIZE_X) * sizeof(float), h_SIZE_X, h_SIZE_Z);
    hipMemcpy3D(&copyParams);
    copyParams.dstArray = cuArray1;
    copyParams.srcPtr = make_hipPitchedPtr(d_V+h_sYZX, (h_SIZE_X) * sizeof(float), h_SIZE_X, h_SIZE_Z);
    hipMemcpy3D(&copyParams);

    // -- 绑定纹理对象
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray; // 注意，类型变了！

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode   = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    // -- 创建纹理对象
    resDesc.res.array.array = cuArray0;  
    hipCreateTextureObject(&texObj0, &resDesc, &texDesc, NULL);
    hipCreateSurfaceObject(&surfObj0, &resDesc);
    resDesc.res.array.array = cuArray1;
    hipCreateTextureObject(&texObj1, &resDesc, &texDesc, NULL);
    hipCreateSurfaceObject(&surfObj1, &resDesc);

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_texObj0), &texObj0, sizeof(hipTextureObject_t));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_texObj1), &texObj1, sizeof(hipTextureObject_t));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_surfObj0), &surfObj0, sizeof(hipSurfaceObject_t));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_surfObj1), &surfObj1, sizeof(hipSurfaceObject_t));


}

void copy_cudaarray_to_vtp1() {
    hipExtent extent = make_hipExtent(h_SIZE_X, h_SIZE_Z, h_SIZE_Y);
    hipMemcpy3DParms copyParams = {0};
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;

    copyParams.srcArray = cuArray0;
    copyParams.dstPtr = make_hipPitchedPtr(d_V_tp1, (h_SIZE_X) * sizeof(float), h_SIZE_X, h_SIZE_Z);
    hipMemcpy3D(&copyParams);
 
    copyParams.srcArray = cuArray1;
    copyParams.dstPtr = make_hipPitchedPtr(d_V_tp1+h_sYZX, (h_SIZE_X) * sizeof(float), h_SIZE_X, h_SIZE_Z);
    hipMemcpy3D(&copyParams);

}



// 一轮计算后重置Vtp1
void reset_Vtp1() {
    // hipMemcpy(d_V_tp1, d_V, h_sEYZX * sizeof(float), hipMemcpyDeviceToDevice);

    hipExtent extent = make_hipExtent(h_SIZE_X, h_SIZE_Z, h_SIZE_Y);
    hipMemcpy3DParms copyParams = {0};
    copyParams.dstArray = cuArray0;
    copyParams.srcPtr = make_hipPitchedPtr(d_V, (h_SIZE_X) * sizeof(float), h_SIZE_X, h_SIZE_Z);
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    hipMemcpy3D(&copyParams);

    copyParams.dstArray = cuArray1;
    copyParams.srcPtr = make_hipPitchedPtr(d_V+h_sYZX, (h_SIZE_X) * sizeof(float), h_SIZE_X, h_SIZE_Z);
    hipMemcpy3D(&copyParams);

    printf("reset_Vtp1 is done\n");
}




// 查表函数
__device__ float lookup_V(float X, float Y, float Z, int E) {
    // int E_int = E;
    // int X_down = (int)floorf((X - d_MIN_X) * d_SCALE_TO_INT_X);
    // int Y_down = (int)floorf((Y - d_MIN_Y) * d_SCALE_TO_INT_Y);
    // int Z_down = (int)floorf((Z - d_MIN_Z) * d_SCALE_TO_INT_Z);
    // int X_up   = fminf(X_down + 1, d_SIZE_X - 1);
    // int Y_up   = fminf(Y_down + 1, d_SIZE_Y - 1);
    // int Z_up   = fminf(Z_down + 1, d_SIZE_Z - 1);

    // // float dx   = (X - d_d_X[X_down]) * d_SCALE_TO_INT_X;
    // // float dy   = (Y - d_d_Y[Y_down]) * d_SCALE_TO_INT_Y; 
    // // float dz   = (Z - d_d_Z[Z_down]) * d_SCALE_TO_INT_Z;

    // float dx = (X - d_MIN_X) * d_SCALE_TO_INT_X - X_down;
    // float dy = (Y - d_MIN_Y) * d_SCALE_TO_INT_Y - Y_down;
    // float dz = (Z - d_MIN_Z) * d_SCALE_TO_INT_Z - Z_down;

    // float res = (1 - dx) * (1 - dy) * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_down, Z_down, X_down)] + 
    //             dx * (1 - dy) * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_down, Z_down, X_up)] + 
    //             (1 - dx) * dy * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_up, Z_down, X_down)] + 
    //             dx * dy * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_up, Z_down, X_up)] + 
    //             (1 - dx) * (1 - dy) * dz * d_d_V_tp1[IDX_V(E_int, Y_down, Z_up, X_down)] + 
    //             dx * (1 - dy) * dz * d_d_V_tp1[IDX_V(E_int, Y_down, Z_up, X_up)] + 
    //             (1 - dx) * dy * dz * d_d_V_tp1[IDX_V(E_int, Y_up, Z_up, X_down)] + 
    //             dx * dy * dz * d_d_V_tp1[IDX_V(E_int, Y_up, Z_up, X_up)];

    float X1 = (X - d_MIN_X) * d_SCALE_TO_INT_X + 0.5f;
    float Y1 = (Y - d_MIN_Y) * d_SCALE_TO_INT_Y + 0.5f;
    float Z1 = (Z - d_MIN_Z) * d_SCALE_TO_INT_Z + 0.5f;
    float res = 0;
    if (E == 0) {
        res = tex3D<float>(d_texObj0, X1, Z1, Y1);
    } else {
        res = tex3D<float>(d_texObj1, X1, Z1, Y1);
    }
    return res;              

}



// 设备函数实现
__device__ float monte_carlo_simulation(float XmW, float Y_tp1, float Z_tp1, int E_tp1, float P_tau_tp1, float P_tau_gep_tp1, float l, hiprandStatePhilox4_32_10_t * rng_states, int idx) {
    float d_temp = 0.0f;
    hiprandStatePhilox4_32_10_t s = rng_states[idx];
    
    // 预计算常用值
    const float exp_term = expf((d_MU - l - 0.5f * d_SIGMA * d_SIGMA) * d_DELTA_T);
    const float sqrt_delta_t = sqrtf(d_DELTA_T);
    const float discount_factor = expf(-d_R * d_DELTA_T);
    
    // Monte Carlo 模拟
    for (int i = 0; i < d_MOTECALO_NUMS; i++) {
        // 生成随机数
        float random = hiprand_normal(&s);
        
        // d_temp += 1000 * random;

        // 计算 X(t+1)
        float X_tp1 = XmW * exp_term * expf(d_SIGMA * sqrt_delta_t * random);
        X_tp1 = fminf(X_tp1, d_MAX_X);
        
        // 查找值函数
        float V_tp1 = lookup_V(X_tp1, Y_tp1, Z_tp1, E_tp1);
        
        // 累加结果
        d_temp += discount_factor * (P_tau_tp1 * fmaxf(X_tp1, Y_tp1) + 
                                   P_tau_gep_tp1 * V_tp1);
    }

    rng_states[idx] = s;
    
    return d_temp ;
}



// XYZEW kernel 实现
__global__ void WEYZX_kernel(int offset, int t, hiprandStatePhilox4_32_10_t *rng_states, float l, float a3, float P_tau_gep_tp1) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if (idx >= d_sWEYZX) return;

    // 计算索引
    int index_w = idx / d_sEYZX;
    int remainder = idx % d_sEYZX;
    int index_e = remainder / d_sYZX;
    remainder = remainder % d_sYZX;
    int index_y = remainder / d_sZX;
    remainder = remainder % d_sZX;
    int index_z = remainder / d_sX;
    int index_x = remainder % d_sX;

    // 获取值
    float X = d_d_X[index_x];
    float Y = d_d_Y[index_y];
    float Z = d_d_Z[index_z];
    int E = d_d_E[index_e];
    float W = d_d_W[index_w];

    if (W > Y) return;

    float min_ZYt = fminf(Z, Y);
    float Y_tp1, Z_tp1;

    int E_tp1 = 1 * (E + W == 0);

    // 优化
    // // ---------- 预先算好共用量 ----------
    const float invX  = __frcp_rn(X);               // 1/X  (更省时钟)
    const float XmW   = fmaxf(X - W, 0.0f);         // max(X-W,0)
    const bool  wz    = (W == 0);
    const bool  ez    = (E_tp1 == 0);
    const bool  wle   = (W <= min_ZYt);

    // ---------- path-specific候选值 ----------
    const float Y00 = fmaxf((1.0f + d_A2) * Y, XmW);          // W==0 && E==0
    const float Z00 = a3 * fmaxf((1.0f + d_A2) * Y, XmW);

    const float Y01 = fmaxf(Y, XmW);          // W==0 && E>0
    const float Z01 = fmaxf(Z, a3 * XmW);

    const float Y10 = fmaxf(Y - W, XmW);                     // W>0 && W<=min_ZYt
    const float Z10 = fmaxf(Z, a3 * XmW);
          
    const float Y11 = fminf(Y - W, Y * invX * XmW);  // W>0 && W>min_ZYt
    const float Z11 = fmaxf(Z * invX * XmW, a3 * XmW);

    // ---------- 4 个掩码 ----------
    const float m00 =  wz &  ez;          // W==0 &&  E==0
    const float m01 =  wz & !ez;          // W==0 &&  E>0
    const float m10 = !wz &  wle;         // W>0 &&  W<=min_ZYt
    const float m11 = !wz & !wle;         // W>0 &&  W> min_ZYt

    // ---------- 混合得到最终结果 ----------
    Y_tp1 = m00 * Y00 + m01 * Y01 + m10 * Y10 + m11 * Y11 * (X != 0); 
    Y_tp1 = fmaxf(fminf(Y_tp1, d_MAX_Y), d_MIN_Y);
    Z_tp1 = m00 * Z00 + m01 * Z01 + m10 * Z10 + m11 * Z11 * (X != 0); 
    Z_tp1 = fmaxf(fminf(Z_tp1, d_MAX_Z), d_MIN_Z);



    float P_tau_tp1 = 1 - P_tau_gep_tp1;
    // //Monte Carlo 模拟
    float d_temp = monte_carlo_simulation(
        XmW, Y_tp1, Z_tp1, E_tp1,
        P_tau_tp1, P_tau_gep_tp1,
        l, rng_states, idx
    );

    // 优化代码
    // ─── 仅用 3 条浮点指令 + 1 条乘 fWt *= (t != 0) ──────────
    float fWt = W - d_A1 * fmaxf(W - min_ZYt, 0.0f);   // ← 已同时覆盖两种情况
    fWt       *= (t != 0);                           // t==0 → 置 0

    
    // 存储结果
    d_d_results[idx] = d_temp / d_MOTECALO_NUMS + fWt;
    // d_d_results[idx] = d_temp;
}

// V_tp1 kernel 实现
__global__ void V_tp1_kernel(int offset, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if (idx >= d_sEYZX) return;

    // 计算索引
    int index_e = idx / d_sYZX;
    int remainder = idx % d_sYZX;
    int index_y = remainder / d_sZX;
    remainder = remainder % d_sZX;
    int index_z = remainder / d_sX;
    int index_x = remainder % d_sX;

    float X = d_d_X[index_x];
    float Y = d_d_Y[index_y];
    float Z = d_d_Z[index_z];
    // int E = d_d_E[index_e];

    int W_index = IDX_V(index_e, index_y, index_z, index_x);
    float max_w = d_d_results[W_index];

    if (t == 0) {
        // d_d_V_tp1[idx] = max_w;//对应着d_results[index_x, index_y, index_z, index_e, 0]
        if (index_e == 0) {
            surf3Dwrite(max_w, d_surfObj0, index_x * sizeof(float), index_z, index_y);
        } else if (index_e == 1) {
            surf3Dwrite(max_w, d_surfObj1, index_x * sizeof(float), index_z, index_y);
        } 
        return;
    }

    // 查找最大值
    for (int i = 0; i < d_SIZE_W; i++) {
        if (Y >= d_d_W[i]) {
            float current = d_d_results[W_index + i*d_sEYZX];
            if (current > max_w) {
                max_w = current;
            }
        }
    }

    float temp = fmaxf(fmaxf(Y - d_A1 * (Y - fminf(Z, Y)), X), max_w);
    // d_d_V_tp1[idx] = temp;

    if (index_e == 0) {
        surf3Dwrite(temp, d_surfObj0, index_x * sizeof(float), index_z, index_y);
    } else if (index_e == 1) {
        surf3Dwrite(temp, d_surfObj1, index_x * sizeof(float), index_z, index_y);
    } 
    // d_d_V_tp1[idx] = fmaxf(fmaxf(Y - d_A1 * fmaxf((Y - fminf(Z, Y)), 0.0f), X), max_w) * (X != 0);
}

//used for test function 
__global__ void test_array_kernel(hipTextureObject_t texObj0, hipTextureObject_t texObj1) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= d_sWEYZX) return;

    d_d_results[idx] = 0;

    // 计算索引
    int index_w = idx / d_sEYZX;
    int remainder = idx % d_sEYZX;
    int index_e = remainder / d_sYZX;
    remainder = remainder % d_sYZX;
    int index_y = remainder / d_sZX;
    remainder = remainder % d_sZX;
    int index_z = remainder / d_sX;
    int index_x = remainder % d_sX;

    float X = d_d_X[index_x];
    float Y = d_d_Y[index_y];
    float Z = d_d_Z[index_z];
    int   E = d_d_E[index_e];
    float W = d_d_W[index_w];

    if (W > 0) return;

    // X = d_MAX_X/2;
    // Y = d_MAX_Y/2;
    // Z = d_MAX_Z/2;


    int E_int = E;
    int X_down = (int)floorf((X - d_MIN_X) * d_SCALE_TO_INT_X);
    int Y_down = (int)floorf((Y - d_MIN_Y) * d_SCALE_TO_INT_Y);
    int Z_down = (int)floorf((Z - d_MIN_Z) * d_SCALE_TO_INT_Z);
    int X_up   = fminf(X_down + 1, d_SIZE_X - 1);
    int Y_up   = fminf(Y_down + 1, d_SIZE_Y - 1);
    int Z_up   = fminf(Z_down + 1, d_SIZE_Z - 1);

    float dx   = (X - d_d_X[X_down]) * d_SCALE_TO_INT_X;
    float dy   = (Y - d_d_Y[Y_down]) * d_SCALE_TO_INT_Y; 
    float dz   = (Z - d_d_Z[Z_down]) * d_SCALE_TO_INT_Z;

    // float dx   = 1.0f;
    // float dy   = 1.0f; 
    // float dz   = 1.0f;

    // float dx = (X - d_MIN_X) * d_SCALE_TO_INT_X - X_down;
    // float dy = (Y - d_MIN_Y) * d_SCALE_TO_INT_Y - Y_down;
    // float dz = (Z - d_MIN_Z) * d_SCALE_TO_INT_Z - Z_down;

    float res = (1 - dx) * (1 - dy) * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_down, Z_down, X_down)] + 
                dx * (1 - dy) * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_down, Z_down, X_up)] + 
                (1 - dx) * dy * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_up, Z_down, X_down)] + 
                dx * dy * (1 - dz) * d_d_V_tp1[IDX_V(E_int, Y_up, Z_down, X_up)] + 
                (1 - dx) * (1 - dy) * dz * d_d_V_tp1[IDX_V(E_int, Y_down, Z_up, X_down)] + 
                dx * (1 - dy) * dz * d_d_V_tp1[IDX_V(E_int, Y_down, Z_up, X_up)] + 
                (1 - dx) * dy * dz * d_d_V_tp1[IDX_V(E_int, Y_up, Z_up, X_down)] + 
                dx * dy * dz * d_d_V_tp1[IDX_V(E_int, Y_up, Z_up, X_up)];


    // float X1 = fminf((X - d_MIN_X) * d_SCALE_TO_INT_X, d_SIZE_X - 1) + 0.5f;
    // float Y1 = fminf((Y - d_MIN_Y) * d_SCALE_TO_INT_Y, d_SIZE_Y - 1) + 0.5f;
    // float Z1 = fminf((Z - d_MIN_Z) * d_SCALE_TO_INT_Z, d_SIZE_Z - 1) + 0.5f;

    float X1 = index_x + 0.5f;
    float Y1 = index_y + 0.5f;
    float Z1 = index_z + 0.5f;

    float res2;
    if (E_int == 0) {
        res2 = tex3D<float>(texObj0, X1, Z1, Y1);
    } else {
        res2 = tex3D<float>(texObj1, X1, Z1, Y1);
    }

    d_d_results[idx] = res2 - res;

}




float compute_l(float l, float * trans_tau_d, int T) {
    float a3 = 1.00/(T/h_P);

    // 这一段后续优化为宏
    // MIN_XYZ, h_INITIAL_INVESTMENT, SCALE_TO_INT_X, SCALE_TO_INT_Y, SCALE_TO_INT_Z, SIZE_Z
    int X_index_1= (int)floorf((h_INITIAL_INVESTMENT - h_MIN_X) * h_SCALE_TO_INT_X);
    int X_index_2=(int)fminf(X_index_1 + 1, h_SIZE_X - 1);
    float delta_x = (h_INITIAL_INVESTMENT - h_MIN_X) * h_SCALE_TO_INT_X - X_index_1;

    int Y_index_1 = (int)floorf((h_INITIAL_INVESTMENT - h_MIN_Y) * h_SCALE_TO_INT_Y);
    int Y_index_2 = (int)fminf(Y_index_1 + 1, h_SIZE_Y - 1);
    float delta_y = (h_INITIAL_INVESTMENT - h_MIN_Y) * h_SCALE_TO_INT_Y - Y_index_1;

    int Z_index_1 = (int)floorf((a3 * h_INITIAL_INVESTMENT - h_MIN_Z) * h_SCALE_TO_INT_Z);
    float delta_z = (a3 * h_INITIAL_INVESTMENT - h_MIN_Z) * h_SCALE_TO_INT_Z - Z_index_1;
    int Z_index_2 = (int)fminf(Z_index_1 + 1, h_SIZE_Z - 1);

    int index1 = h_IDX_V(0, Y_index_1, Z_index_1, X_index_1);
    int index2 = h_IDX_V(0, Y_index_1, Z_index_1, X_index_2);   
    int index3 = h_IDX_V(0, Y_index_2, Z_index_1, X_index_1);
    int index4 = h_IDX_V(0, Y_index_1, Z_index_2, X_index_1);
    int index5 = h_IDX_V(0, Y_index_2, Z_index_1, X_index_2);
    int index6 = h_IDX_V(0, Y_index_1, Z_index_2, X_index_2);
    int index7 = h_IDX_V(0, Y_index_2, Z_index_2, X_index_1);
    int index8 = h_IDX_V(0, Y_index_2, Z_index_2, X_index_2);


    // 设置随机数生成器
    init_random_state();

    // 设置block和grid
    dim3 block(896);
    dim3 grid((h_sWEYZX + block.x - 1) / block.x);

    dim3 block2(1024);
    dim3 grid2((h_sEYZX + block2.x - 1) / block2.x);
    for (int t = T-1; t >= 0; t--) {
        float P_tau_t = trans_tau_d[t];
        
        // 计算V(t)
        // t = -1;
        WEYZX_kernel<<<grid, block>>>(0, t, d_rng_states, l, a3, P_tau_t);
        CUDA_CHECK(hipGetLastError());     // launch
        CUDA_CHECK(hipDeviceSynchronize()); // runtime

        // 计算W的最大值
        V_tp1_kernel<<<grid2, block2>>>(0, t);
        CUDA_CHECK(hipGetLastError());     // launch
        CUDA_CHECK(hipDeviceSynchronize()); // runtime


    }

    copy_cudaarray_to_vtp1();

    float out1, out2, out3, out4, out5, out6, out7, out8;
    hipMemcpy(&out1, &d_V_tp1[index1], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out2, &d_V_tp1[index2], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out3, &d_V_tp1[index3], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out4, &d_V_tp1[index4], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out5, &d_V_tp1[index5], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out6, &d_V_tp1[index6], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out7, &d_V_tp1[index7], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out8, &d_V_tp1[index8], sizeof(float), hipMemcpyDeviceToHost);


    float output = (1-delta_x) * (1-delta_y) * (1-delta_z) * out1
                 + delta_x * (1-delta_y) * (1-delta_z) * out2
                 + (1-delta_x) * delta_y * (1-delta_z) * out3
                 + (1-delta_x) * (1-delta_y) * delta_z * out4
                 + delta_x * delta_y * (1-delta_z) * out5
                 + delta_x * (1-delta_y) * delta_z * out6
                 + (1-delta_x) * delta_y * delta_z * out7
                 + delta_x * delta_y * delta_z * out8;


    float final_X_1, final_X_2, final_Y_1, final_Y_2, final_Z_1, final_Z_2;
    hipMemcpy(&final_X_1, &d_X[X_index_1], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_X_2, &d_X[X_index_2], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_Y_1, &d_Y[Y_index_1], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_Y_2, &d_Y[Y_index_2], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_Z_1, &d_Z[Z_index_1], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_Z_2, &d_Z[Z_index_2], sizeof(float), hipMemcpyDeviceToHost);

    // float *h_Z = (float *)malloc(h_SIZE_Z * sizeof(float));
    // hipMemcpy(h_Z, d_Z, h_SIZE_Z * sizeof(float), hipMemcpyDeviceToHost);
    // for(int i = 0 ; i< h_SIZE_Z;i++){
    // printf("Z = %f\n", h_Z[i]);}

    printf("X_index_1 = %d, X_index_2 = %d, Y_index_1 = %d, Y_index_2 = %d, Z_index_1 = %d, Z_index_2 = %d\n", 
            X_index_1, X_index_2, Y_index_1, Y_index_2, Z_index_1, Z_index_2);
    printf("1/2---对应的账户值是：%f, %f, %f, %f, %f, %f\n", 
            final_X_1, final_X_2, final_Y_1, final_Y_2, final_Z_1, final_Z_2);
    printf("相应的delta值是：%f, %f, %f\n", delta_x, delta_y, delta_z);
    printf("out1 = %f, out2 = %f, out3 = %f, out4 = %f, out5 = %f, out6 = %f, out7 = %f, out8 = %f, output = %f\n", 
            out1, out2, out3, out4, out5, out6, out7, out8, output);


    return output;
}

