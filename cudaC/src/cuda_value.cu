#include "hip/hip_runtime.h"
#include "cuda_value.h"


// 生成随机数
__global__ void setup(hiprandStatePhilox4_32_10_t *state, unsigned long seed, int PATHS)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= PATHS) return;
    /* sequence=tid, offset=0 → 线程独立子流 */
    hiprand_init(seed, tid, 0, &state[tid]);
}

void init_random_state(hiprandStatePhilox4_32_10_t *d_state, unsigned long seed, int PATHS){
    hipMalloc(&d_state,  PATHS*sizeof(*d_state));
    setup<<<(PATHS+1023)/1024,1024>>>(d_state, seed, PATHS);
    hipDeviceSynchronize();
}



// 在 cuda_value.cu 文件开头定义全局变量
// __device__ __host__ float *d_X, *d_Y, *d_Z, *d_W, *d_V, *d_V_tp1;
// __device__ __host__ int *d_E;

__host__ __device__ int IDX_V(int x, int y, int z, int e) {
    // if(x*sYZEW + y*sZEW + z*sEW + e*sW + e >= SIZE_X * SIZE_Y * SIZE_Z * SIZE_E) {
    //     printf("Error: Index out of bounds\n");
    //     exit(1);
    // }
    
    return (x*sYZE + y*sZE + z*sE + e);
}

__host__ int h_IDX_V(int x, int y, int z, int e){
    return (x*h_sYZE + y*h_sZE + z*h_sE + e);
}

// cuda_value.cu
int h_MIN_X, h_MIN_Y, h_MIN_Z, h_MIN_W;
int h_MAX_X, h_MAX_Y, h_MAX_Z, h_MAX_W;
int h_SIZE_X, h_SIZE_Y, h_SIZE_Z, h_SIZE_E, h_SIZE_W;
int h_sXYZEW, h_sYZEW, h_sZEW, h_sEW, h_sW;
int h_sXYZE, h_sYZE, h_sZE, h_sE;
float h_SCALE_TO_INT_X, h_SCALE_TO_INT_Y, h_SCALE_TO_INT_Z;

float *d_X, *d_Y, *d_Z, *d_W, *d_V, *d_V_tp1, *d_results;
int *d_E;

void init_global_config(
    int min_X, int max_X, int size_X,
    int min_Y, int max_Y, int size_Y,
    int min_Z, int max_Z, int size_Z,
    int min_E, int max_E, int size_E,
    int min_W, int max_W, int size_W
){
    h_MIN_X = min_X;
    h_MAX_X = max_X;
    h_SIZE_X = size_X;
    h_MIN_Y = min_Y;
    h_MAX_Y = max_Y;
    h_SIZE_Y = size_Y;
    h_MIN_Z = min_Z;
    h_MAX_Z = max_Z;
    h_SIZE_Z = size_Z;
    h_SIZE_E = size_E;
    h_MIN_W = min_W;
    h_MAX_W = max_W;
    h_SIZE_W = size_W;

    h_sXYZEW = size_X * size_Y * size_Z * size_E * size_W;
    h_sYZEW = size_Y * size_Z * size_E * size_W;
    h_sZEW = size_Z * size_E * size_W;
    h_sEW = size_E * size_W;
    h_sW = size_W;

    h_sXYZE = size_X * size_Y * size_Z * size_E;   
    h_sYZE = size_Y * size_Z * size_E;
    h_sZE = size_Z * size_E;
    h_sE = size_E;

    h_SCALE_TO_INT_X = (float)(size_X-1) / (max_X - min_X);
    h_SCALE_TO_INT_Y = (float)(size_Y-1) / (max_Y - min_Y);
    h_SCALE_TO_INT_Z = (float)(size_Z-1) / (max_Z - min_Z);

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_X), &h_MIN_X, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_X), &h_MAX_X, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_X), &h_SIZE_X, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_Y), &h_MIN_Y, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_Y), &h_MAX_Y, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_Y), &h_SIZE_Y, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_Z), &h_MIN_Z, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_Z), &h_MAX_Z, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_Z), &h_SIZE_Z, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MIN_W), &h_MIN_W, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_MAX_W), &h_MAX_W, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SIZE_W), &h_SIZE_W, sizeof(int));

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sXYZEW), &h_sXYZEW, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sYZEW), &h_sYZEW, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sZEW), &h_sZEW, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sEW), &h_sEW, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sW), &h_sW, sizeof(int));

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sXYZE), &h_sXYZE, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sYZE), &h_sYZE, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sZE), &h_sZE, sizeof(int));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_sE), &h_sE, sizeof(int));

    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SCALE_TO_INT_X), &h_SCALE_TO_INT_X, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SCALE_TO_INT_Y), &h_SCALE_TO_INT_Y, sizeof(float));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_SCALE_TO_INT_Z), &h_SCALE_TO_INT_Z, sizeof(float));


    // 初始化XYZEW_V
    float *h_X = (float *)malloc(h_SIZE_X * sizeof(float));
    float *h_Y = (float *)malloc(h_SIZE_Y * sizeof(float));
    float *h_Z = (float *)malloc(h_SIZE_Z * sizeof(float));
    int   *h_E = (int   *)malloc(h_SIZE_E * sizeof(int));
    float *h_W = (float *)malloc(h_SIZE_W * sizeof(float));
    float *h_V = (float *)malloc(h_sXYZE * sizeof(float));
    
    for (int i = 0; i < h_SIZE_X; i++) {
        h_X[i] = h_MIN_X + (h_MAX_X - h_MIN_X) * i / (h_SIZE_X - 1);
    }
    for (int i = 0; i < h_SIZE_Y; i++) {
        h_Y[i] = h_MIN_Y + (h_MAX_Y - h_MIN_Y) * i / (h_SIZE_Y - 1);
    }
    for (int i = 0; i < h_SIZE_Z; i++) {
        h_Z[i] = h_MIN_Z + (h_MAX_Z - h_MIN_Z) * i / (h_SIZE_Z - 1);
    }
    for (int i = 0; i < h_SIZE_E; i++) {
        h_E[i] = i;
    }
    for (int i = 0; i < h_SIZE_W; i++) {
        h_W[i] = h_MIN_W + (h_MAX_W - h_MIN_W) * i / (h_SIZE_W - 1);
    }

    // 初始化 V 数组
    printf("Initializing V array...\n");
    for (int x = 0; x < h_SIZE_X; x++) {
        for (int y = 0; y < h_SIZE_Y; y++) {
            for (int z = 0; z < h_SIZE_Z; z++) {
                float min_ZY = fminf(h_Z[z], h_Y[y]);
                float term = (h_Y[y] <= min_ZY) ? 
                            h_Y[y] : 
                            h_Y[y] - A1 * (h_Y[y] - min_ZY);
                float result = fmaxf(h_X[x], term);
                
                // 对 E 的两个维度都赋值
                h_V[h_IDX_V(x, y, z, 0)] = result;
                h_V[h_IDX_V(x, y, z, 1)] = result;
            }
        }
    }

    printf("V array initialized\n");

    // 分配设备内存
    hipMalloc(&d_X, h_SIZE_X * sizeof(float));
    hipMalloc(&d_Y, h_SIZE_Y * sizeof(float));
    hipMalloc(&d_Z, h_SIZE_Z * sizeof(float)); 
    hipMalloc(&d_E, h_SIZE_E * sizeof(int));
    hipMalloc(&d_W, h_SIZE_W * sizeof(float));
    hipMalloc(&d_V, h_sXYZE * sizeof(float));
    hipMalloc(&d_V_tp1, h_sXYZE * sizeof(float));
    hipMalloc(&d_results, h_sXYZEW * sizeof(float));
    printf("cudamalloc done\n");
    // 检查内存分配是否成功
    // if (!d_X || !d_Y || !d_Z || !d_E || !d_W || !d_V || !d_V_tp1 || !d_results) {
    //     printf("Error: Failed to allocate device memory\n");
    //     clean_global_XYZEW_V();
    //     exit(1);
    // }

    

    // 复制数据到设备
    hipMemcpy(d_X, h_X, h_SIZE_X * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, h_SIZE_Y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, h_Z, h_SIZE_Z * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_E, h_E, h_SIZE_E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_W, h_W, h_SIZE_W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, h_sXYZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V_tp1, h_V, h_sXYZE * sizeof(float), hipMemcpyHostToDevice);
    

    // 将主机端指针值复制到设备端全局变量
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_X), &d_X, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_Y), &d_Y, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_Z), &d_Z, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_E), &d_E, sizeof(int*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_W), &d_W, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_V), &d_V, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_V_tp1), &d_V_tp1, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_results), &d_results, sizeof(float*));

    printf("init_global_XYZEW_V is done\n");



    // 释放主机内存
    free(h_X);
    free(h_Y);
    free(h_Z);
    free(h_E);
    free(h_W);
    free(h_V);

}

void init_global_XYZEW_V() {
    // 分配主机内存
    float *h_X = (float *)malloc(SIZE_X * sizeof(float));
    float *h_Y = (float *)malloc(SIZE_Y * sizeof(float));
    float *h_Z = (float *)malloc(SIZE_Z * sizeof(float));
    int   *h_E = (int   *)malloc(SIZE_E * sizeof(int));
    float *h_W = (float *)malloc(SIZE_W * sizeof(float));
    float *h_V = (float *)malloc(SIZE_X * SIZE_Y * SIZE_Z * SIZE_E *sizeof(float));
    
    // 初始化 X, Y, Z, W 数组
    for (int i = 0; i < SIZE_X; i++) {
        h_X[i] = MIN_XYZ + (MAX_X - MIN_XYZ) * i / (SIZE_X - 1);
    }
    for (int i = 0; i < SIZE_Y; i++) {
        h_Y[i] = MIN_XYZ + (MAX_Y - MIN_XYZ) * i / (SIZE_Y - 1);
    }
    for (int i = 0; i < SIZE_Z; i++) {
        h_Z[i] = MIN_XYZ + (MAX_Z - MIN_XYZ) * i / (SIZE_Z - 1);
    }
    for (int i = 0; i < SIZE_E; i++) {
        h_E[i] = i;
    }
    for (int i = 0; i < SIZE_W; i++) {
        h_W[i] = MIN_XYZ + (MAX_W - MIN_XYZ) * i / (SIZE_W - 1);
    }

    // 初始化 V 数组
    printf("Initializing V array...\n");
    for (int x = 0; x < SIZE_X; x++) {
        for (int y = 0; y < SIZE_Y; y++) {
            for (int z = 0; z < SIZE_Z; z++) {
                float min_ZY = fminf(h_Z[z], h_Y[y]);
                float term = (h_Y[y] <= min_ZY) ? 
                            h_Y[y] : 
                            h_Y[y] - A1 * (h_Y[y] - min_ZY);
                float result = fmaxf(h_X[x], term);
                
                // 对 E 的两个维度都赋值
                h_V[IDX_V(x, y, z, 0)] = result;
                h_V[IDX_V(x, y, z, 1)] = result;
            }
        }
    }

    // 分配设备内存
    hipMalloc(&d_X, SIZE_X * sizeof(float));
    hipMalloc(&d_Y, SIZE_Y * sizeof(float));
    hipMalloc(&d_Z, SIZE_Z * sizeof(float));
    hipMalloc(&d_E, SIZE_E * sizeof(int));
    hipMalloc(&d_W, SIZE_W * sizeof(float));
    hipMalloc(&d_V, SIZE_X * SIZE_Y * SIZE_Z * SIZE_E * sizeof(float));
    hipMalloc(&d_V_tp1, SIZE_X * SIZE_Y * SIZE_Z * SIZE_E * sizeof(float));
    hipMalloc(&d_results, SIZE_X * SIZE_Y * SIZE_Z * SIZE_E * SIZE_W * sizeof(float));

    // 检查内存分配是否成功
    if (!d_X || !d_Y || !d_Z || !d_E || !d_W || !d_V || !d_V_tp1 || !d_results) {
        printf("Error: Failed to allocate device memory\n");
        clean_global_XYZEW_V();
        exit(1);
    }

    // 复制数据到设备
    hipMemcpy(d_X, h_X, SIZE_X * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Y, h_Y, SIZE_Y * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Z, h_Z, SIZE_Z * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_E, h_E, SIZE_E * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_W, h_W, SIZE_W * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, SIZE_X * SIZE_Y * SIZE_Z * SIZE_E * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_V_tp1, h_V, SIZE_X * SIZE_Y * SIZE_Z * SIZE_E * sizeof(float), hipMemcpyHostToDevice);


    // 将主机端指针值复制到设备端全局变量
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_X), &d_X, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_Y), &d_Y, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_Z), &d_Z, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_E), &d_E, sizeof(int*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_W), &d_W, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_V), &d_V, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_V_tp1), &d_V_tp1, sizeof(float*));
    hipMemcpyToSymbolAsync(HIP_SYMBOL(d_d_results), &d_results, sizeof(float*));

    // 检查内存复制是否成功
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error: %s\n", hipGetErrorString(error));
        clean_global_XYZEW_V();
        exit(1);
    }




    // 释放主机内存
    free(h_X);
    free(h_Y);
    free(h_Z);
    free(h_E);
    free(h_W);
    free(h_V);
}

// 清理函数
void clean_global_XYZEW_V() {
    if (d_X) hipFree(d_X);
    if (d_Y) hipFree(d_Y);
    if (d_Z) hipFree(d_Z);
    if (d_E) hipFree(d_E);
    if (d_W) hipFree(d_W);
    if (d_V) hipFree(d_V);
    if (d_V_tp1) hipFree(d_V_tp1);
    if (d_results) hipFree(d_results);

    d_X = nullptr;
    d_Y = nullptr;
    d_Z = nullptr;
    d_E = nullptr;
    d_W = nullptr;
    d_V = nullptr;
    d_V_tp1 = nullptr;
    d_results = nullptr;

    printf("clean_global_XYZEW_V is done\n");
}   


// 一轮计算后重置Vtp1
void reset_Vtp1() {
    hipMemcpy(d_V_tp1, d_V, sXYZE * sizeof(float), hipMemcpyDeviceToDevice);
}




// 查表函数
__device__ float lookup_V(float X, float Y, float Z, int E) {
    float scale_to_int = (float)SIZE_X / (MAX_X - MIN_XYZ);
    
    int X_int = (int)floorf((X - MIN_XYZ) * scale_to_int);
    int Y_int = (int)floorf((Y - MIN_XYZ) * scale_to_int);
    int Z_int = (int)floorf((Z - MIN_XYZ) * scale_to_int);
    int E_int = E;
    
    return d_d_V_tp1[IDX_V(X_int, Y_int, Z_int, E_int)];
}



// 设备函数实现
__device__ float monte_carlo_simulation(float XmW, float Y_tp1, float Z_tp1, int E_tp1, float P_tau_tp1, float P_tau_gep_tp1, float l, hiprandStatePhilox4_32_10_t * rng_states, int idx) {
    float d_temp = 0.0f;
    hiprandStatePhilox4_32_10_t s = rng_states[idx];
    
    // 预计算常用值
    const float exp_term = expf((MU - l - 0.5f * SIGMA * SIGMA) * DELTA_T);
    const float sqrt_delta_t = sqrtf(DELTA_T);
    const float discount_factor = expf(-R * DELTA_T);
    
    // Monte Carlo 模拟
    for (int i = 0; i < MOTECALO_NUMS; i++) {
        // 生成随机数
        float random = hiprand_normal(&s);
        
        // d_temp += 1000 * random;

        // 计算 X(t+1)
        float X_tp1 = XmW * exp_term * expf(SIGMA * sqrt_delta_t * random);
        X_tp1 = fminf(X_tp1, MAX_X);
        
        // 查找值函数
        float V_tp1 = lookup_V(X_tp1, Y_tp1, Z_tp1, E_tp1);
        
        // 累加结果
        d_temp += discount_factor * (P_tau_tp1 * fmaxf(X_tp1, Y_tp1) + 
                                   P_tau_gep_tp1 * V_tp1);
    }

    rng_states[idx] = s;
    
    return d_temp ;
}

// XYZEW kernel 实现
__global__ void XYZEW_kernel(int offset, int t, hiprandStatePhilox4_32_10_t *rng_states, float l, float a3, float P_tau_gep_tp1) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if (idx >= sXYZEW) return;

    // 计算索引
    int index_x = idx / sYZEW;
    int remainder = idx % sYZEW;
    int index_y = remainder / sZEW;
    remainder = remainder % sZEW;
    int index_z = remainder / sEW;
    remainder = remainder % sEW;
    int index_e = remainder / sW;
    int index_w = remainder % sW;

    // 获取值
    float X = d_d_X[index_x];
    float Y = d_d_Y[index_y];
    float Z = d_d_Z[index_z];
    int E = d_d_E[index_e];
    float W = d_d_W[index_w];

    float min_ZYt = fminf(Z, Y);
    float Y_tp1, Z_tp1;

    int E_tp1 = 1 * (E + W == 0);





    // 优化
    // // ---------- 预先算好共用量 ----------
    const float invX  = __frcp_rn(X);               // 1/X  (更省时钟)
    const float XmW   = fmaxf(X - W, 0.0f);         // max(X-W,0)
    const bool  wz    = (W == 0);
    const bool  ez    = (E_tp1 == 0);
    const bool  wle   = (W <= min_ZYt);

    // ---------- path-specific候选值 ----------
    const float Y00 = (1.0f + A2) * fmaxf(X,        Y);          // W==0 && E==0
    const float Z00 = (1.0f + A2) * fmaxf(a3 * X,   Z);

    const float Y01 =                fmaxf(X,        Y);          // W==0 && E>0
    const float Z01 =                fmaxf(a3 * X,   Z);

    const float Y10 = fmaxf(XmW,      Y - W);                     // W>0 && W<=min_ZYt
    const float Z10 = fmaxf(a3 * XmW, Z);

    const float t111    = fminf(Y - W,   Y * invX * XmW);            // W>0 && W>min_ZYt
    const float Y11 = fmaxf(XmW,      t111);
    const float Z11 = fmaxf(a3 * XmW, Z * invX * XmW);

    // ---------- 4 个掩码 ----------
    const float m00 =  wz &  ez;          // W==0 &&  E==0
    const float m01 =  wz & !ez;          // W==0 &&  E>0
    const float m10 = !wz &  wle;         // W>0 &&  W<=min_ZYt
    const float m11 = !wz & !wle;         // W>0 &&  W> min_ZYt

    // ---------- 混合得到最终结果 ----------
    Y_tp1 = m00 * Y00 + m01 * Y01 + m10 * Y10 + m11 * Y11;
    Z_tp1 = m00 * Z00 + m01 * Z01 + m10 * Z10 + m11 * Z11;

        // P_tau_tp1 = d_P_tau[0] # 这个是P(tau=t+1)时刻的值
        // P_tau_gep_tp1 = d_P_tau[1] # 这个是P(tau>=t+1)时刻的值


    float P_tau_tp1 = 1 - P_tau_gep_tp1;

 

    // //Monte Carlo 模拟
    float d_temp = monte_carlo_simulation(
        XmW, Y_tp1, Z_tp1, E_tp1,
        P_tau_tp1, P_tau_gep_tp1,
        l, rng_states, idx
    );



    // 优化代码
    // ─── 仅用 3 条浮点指令 + 1 条乘 fWt *= (t != 0) ──────────
    float fWt = W - A1 * fmaxf(W - min_ZYt, 0.0f);   // ← 已同时覆盖两种情况
    fWt       *= (t != 0);                           // t==0 → 置 0

    
    // 存储结果
    d_d_results[idx] = d_temp / MOTECALO_NUMS + fWt;
    // d_results[idx] = d_temp;
}

// V_tp1 kernel 实现
__global__ void V_tp1_kernel(int offset, int t) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if (idx >= SIZE_X * SIZE_Y * SIZE_Z * SIZE_E) return;

    // 计算索引
    int index_x = idx / sYZE;
    int remainder = idx % sYZE;
    int index_y = remainder / sZE;
    remainder = remainder % sZE;
    int index_z = remainder / sE;
    // int index_e = remainder % sE;

    float X = d_d_X[index_x];
    float Y = d_d_Y[index_y];
    float Z = d_d_Z[index_z];
    // int E = d_d_E[index_e];

    int W_index = idx * SIZE_W;
    float max_w = d_d_results[W_index];

    if (t == 0) {
        d_d_V_tp1[idx] = max_w;
        return;
    }

    // 查找最大值
    for (int i = 0; i < SIZE_W; i++) {

        if (Y >= d_d_W[i]) {
            float current = d_d_results[W_index + i];
            if (current > max_w) {
                max_w = current;
            }
        }
    }

    d_d_V_tp1[idx] = fmaxf(fmaxf(Y - A1 * fmaxf((Y - fminf(Z, Y)), 0.0f), X), max_w);
}



