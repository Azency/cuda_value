#include "hip/hip_runtime.h"
#include "cuda_value.h"


// 导出到python的函数
extern "C" 
float pycompute_l(float l, float * trans_tau_d, int T) {
    float a3 = 1.00/(T/h_P);

    // 这一段后续优化为宏
    // MIN_XYZ, h_INITIAL_INVESTMENT, SCALE_TO_INT_X, SCALE_TO_INT_Y, SCALE_TO_INT_Z, SIZE_Z
    int X_index = (int)floorf((h_INITIAL_INVESTMENT - h_MIN_X) * h_SCALE_TO_INT_X);
    int Y_index = (int)floorf((h_INITIAL_INVESTMENT - h_MIN_Y) * h_SCALE_TO_INT_Y);
    int Z_index_1 = (int)floorf((a3 * h_INITIAL_INVESTMENT - h_MIN_Z) * h_SCALE_TO_INT_Z);
    float delta_z = (a3 * h_INITIAL_INVESTMENT - h_MIN_Z) * h_SCALE_TO_INT_Z - Z_index_1;
    int Z_index_2 = (int)fminf(Z_index_1 + 1, h_SIZE_Z - 1);


    int index1 = h_IDX_V(X_index, Y_index, Z_index_1, 0);
    int index2 = h_IDX_V(X_index, Y_index, Z_index_2, 0);

    // 设置随机数生成器
    hiprandStatePhilox4_32_10_t* d_rng_states;
    int num_threads = h_sXYZEW;
    hipMalloc(&d_rng_states,  num_threads*sizeof(*d_rng_states));
    setup<<<(num_threads+1023)/1024,1024>>>(d_rng_states, 101, num_threads);

    // 设置block和grid
    dim3 block(512);
    dim3 grid((h_sXYZEW + block.x - 1) / block.x);

    dim3 block2(512);
    dim3 grid2((h_sXYZE + block2.x - 1) / block2.x);
    for (int t = T-1; t >= 0; t--) {
        float P_tau_t = trans_tau_d[t];
        
        // 计算V(t)
        XYZEW_kernel<<<grid, block>>>(0, t, d_rng_states, l, a3, P_tau_t);
        CUDA_CHECK(hipGetLastError());     // launch
        CUDA_CHECK(hipDeviceSynchronize()); // runtime

        // 计算W的最大值
        V_tp1_kernel<<<grid2, block2>>>(0, t);
        CUDA_CHECK(hipGetLastError());     // launch
        CUDA_CHECK(hipDeviceSynchronize()); // runtime


    }

    float out1, out2;
    hipMemcpy(&out1, &d_V_tp1[index1], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out2, &d_V_tp1[index2], sizeof(float), hipMemcpyDeviceToHost);
  
    float output = out1 + (out2 - out1)*delta_z;


    float final_X, final_Y, final_Z_1, final_Z_2;
    hipMemcpy(&final_X, &d_X[X_index], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_Y, &d_Y[Y_index], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_Z_1, &d_Z[Z_index_1], sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&final_Z_2, &d_Z[Z_index_2], sizeof(float), hipMemcpyDeviceToHost);
    
    // printf("index1 = %d, index2 = %d\n", index1, index2);
    printf("out1 = %f, out2 = %f, output = %f\n", out1, out2, output);
    printf("X_index = %d, Y_index = %d, Z_index_1 = %d, Z_index_2 = %d\n", X_index, Y_index, Z_index_1, Z_index_2);
    printf("1/2---对应的账户值是：%f, %f, %f, %f\n", final_X, final_Y, final_Z_1, final_Z_2);


    hipFree(d_rng_states);

    return output;
}



extern "C"
void pyinit_global_XYZEW_V() {
    init_global_XYZEW_V();
}

extern "C"
void pyclean_global_XYZEW_V() {
    clean_global_XYZEW_V();
}

extern "C"
void pyreset_Vtp1() {
    reset_Vtp1();
}

extern "C"
void pyinit_global_config(
    int min_X, int max_X, int size_X,
    int min_Y, int max_Y, int size_Y,
    int min_Z, int max_Z, int size_Z,
    int min_E, int max_E, int size_E,
    int min_W, int max_W, int size_W,
    float a1, float a2, float r, float mu, float sigma, int motecalo_nums, float p, float initial_investment
) {
    init_global_config(
        min_X, max_X, size_X, 
        min_Y, max_Y, size_Y, 
        min_Z, max_Z, size_Z, 
        min_E, max_E, size_E, 
        min_W, max_W, size_W, 
        a1, a2, r, mu, sigma, motecalo_nums, p, initial_investment);
}