#include "hip/hip_runtime.h"
// test_init.cu
#include "cuda_value.h"





std::vector<float> trans_tau_np = {0.95227777, 0.9458399, 0.938519, 0.93016787, 0.92060485, 0.9096251, 0.89702214, 0.88261673, 0.86628806, 0.84799892};



int main() {

    float l = 0.00f;

    init_global_XYZEW_V();

    printf("init_global_XYZEW_V done\n");

    // 设置随机数生成器
    float output = compute_l(l, trans_tau_np);

    std::cout << "output = " << output << std::endl;

    clean_global_XYZEW_V();


    return 0;
}


// 测试随机数 ----------------------------      ------------------------------  start
// #include <hiprand/hiprand_kernel.h>

// __global__ void mc_kernel(hiprandStatePhilox4_32_10_t *state,
//                           float *payoff, int steps)
// {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     hiprandStatePhilox4_32_10_t s = state[tid];

//     float S = 100.f;                       // 例如股票现价
//     float mu = 0.06f, sigma = 0.2f, dt = 1.f/252;
//     // for (int t = 0; t < steps; ++t) {
//     //     float z = hiprand_normal(&s);       // N(0,1)
//     //     S *= __expf((mu - .5f*sigma*sigma)*dt + sigma*sqrtf(dt)*z);
//     // }
//     payoff[tid] = hiprand_normal(&s);   // 欧式看涨

//     state[tid] = s;                        // 写回
// }

// __global__ void setup(hiprandStatePhilox4_32_10_t *state, unsigned long seed, int PATHS)
// {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid >= PATHS) return;
//     /* sequence=tid, offset=0 → 线程独立子流 */
//     hiprand_init(seed, tid, 0, &state[tid]);
// }

// int main() {
//     const int PATHS = 1<<20;
//     hiprandStatePhilox4_32_10_t *d_state;
//     float *d_payoff;
//     hipMalloc(&d_state,  PATHS*sizeof(*d_state));
//     hipMalloc(&d_payoff, PATHS*sizeof(float));

//     setup<<<PATHS/256,256>>>(d_state, 101);
//     mc_kernel<<<PATHS/256,256>>>(d_state, d_payoff, /*steps=*/252);

//     float h_payoff[PATHS];
//     hipMemcpy(h_payoff, d_payoff, PATHS*sizeof(float), hipMemcpyDeviceToHost);
//     printf("h_payoff = %f\n", h_payoff[14]);

//     // 取均值 …
// }


















// 测试随机数 ----------------------------      ------------------------------  end





















// 测试初始化 ----------------------------      ------------------------------  start







    // // 打印数组内容的辅助函数
    // void print_array(const char* name, float* arr, int size) {
    //     std::cout << name << " = [";
    //     for (int i = 0; i < std::min(size, 5); i++) {
    //         std::cout << std::fixed << std::setprecision(2) << arr[i] << " ";
    //     }
    //     if (size > 5) {
    //         std::cout << "... ";
    //         for (int i = size - 5; i < size; i++) {
    //             std::cout << std::fixed << std::setprecision(2) << arr[i] << " ";
    //         }
    //     }
    //     std::cout << "]" << std::endl;
    // }

    // // 验证数组初始化是否正确
    // bool verify_initialization() {
    //     // 分配主机内存用于验证
    //     float *h_X = new float[SIZE_X];
    //     float *h_Y = new float[SIZE_Y];
    //     float *h_Z = new float[SIZE_Z];
    //     int *h_E = new int[SIZE_E];
    //     float *h_W = new float[SIZE_W];
    //     float *h_V = new float[SIZE_X * SIZE_Y * SIZE_Z * SIZE_E];
    //     float *h_V_tp1 = new float[SIZE_X * SIZE_Y * SIZE_Z * SIZE_E];

    //     // 从设备复制数据到主机
    //     hipMemcpy(h_X, d_X, SIZE_X * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(h_Y, d_Y, SIZE_Y * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(h_Z, d_Z, SIZE_Z * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(h_E, d_E, SIZE_E * sizeof(int), hipMemcpyDeviceToHost);
    //     hipMemcpy(h_W, d_W, SIZE_W * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(h_V, d_V, SIZE_X * SIZE_Y * SIZE_Z * SIZE_E * sizeof(float), hipMemcpyDeviceToHost);
    //     hipMemcpy(h_V_tp1, d_V_tp1, SIZE_X * SIZE_Y * SIZE_Z * SIZE_E * sizeof(float), hipMemcpyDeviceToHost);

    //     // 验证数据
    //     bool success = true;

    //     // 验证 X 数组
    //     for (int i = 0; i < SIZE_X; i++) {
    //         float expected = MIN_XYZ + (MAX_X - MIN_XYZ) * i / (SIZE_X - 1);
    //         if (fabs(h_X[i] - expected) > 1e-6) {
    //             std::cout << "X array verification failed at index " << i 
    //                     << ": expected " << expected << ", got " << h_X[i] << std::endl;
    //             success = false;
    //             break;
    //         }
    //     }

    //     // 验证 E 数组
    //     for (int i = 0; i < SIZE_E; i++) {
    //         if (h_E[i] != i) {
    //             std::cout << "E array verification failed at index " << i 
    //                     << ": expected " << i << ", got " << h_E[i] << std::endl;
    //             success = false;
    //             break;
    //         }
    //     }

    //     // 验证 V 数组的特定位置
    //     for (int x = 0; x < std::min(SIZE_X, 2); x++) {
    //         for (int y = 0; y < std::min(SIZE_Y, 2); y++) {
    //             for (int z = 0; z < std::min(SIZE_Z, 2); z++) {
    //                 for (int e = 0; e < SIZE_E; e++) {
    //                     float min_ZY = fminf(h_Z[z], h_Y[y]);
    //                     float term = (h_Y[y] <= min_ZY) ? 
    //                                 h_Y[y] : 
    //                                 h_Y[y] - A1 * (h_Y[y] - min_ZY);
    //                     float expected = fmaxf(h_X[x], term);
    //                     float actual = h_V[IDX_V(x, y, z, e)];
                        
    //                     if (fabs(actual - expected) > 1e-6) {
    //                         std::cout << "V array verification failed at (" << x << "," << y << "," 
    //                                 << z << "," << e << "): expected " << expected 
    //                                 << ", got " << actual << std::endl;
    //                         success = false;
    //                         break;
    //                     }
    //                 }
    //             }
    //         }
    //     }

    //     // 验证 V_tp1 是否和 V 一致
    //     for (int i = 0; i < SIZE_X * SIZE_Y * SIZE_Z * SIZE_E; i++) {
    //         if (fabs(h_V[i] - h_V_tp1[i]) > 1e-6) {
    //             std::cout << "V_tp1 array verification failed at index " << i
    //                     << ": expected " << h_V[i] << ", got " << h_V_tp1[i] << std::endl;
    //             success = false;
    //             break;
    //         }
    //     }

    //     // 打印部分内容
    //     print_array("X", h_X, SIZE_X);
    //     print_array("Y", h_Y, SIZE_Y);
    //     print_array("Z", h_Z, SIZE_Z);
    //     print_array("E", reinterpret_cast<float*>(h_E), SIZE_E);
    //     print_array("W", h_W, SIZE_W);

    //     // 释放主机内存
    //     delete[] h_X;
    //     delete[] h_Y;
    //     delete[] h_Z;
    //     delete[] h_E;
    //     delete[] h_W;
    //     delete[] h_V;
    //     delete[] h_V_tp1;

    //     return success;
    // }

    // int main() {
    //     std::cout << "Testing init_global_XYZEW_V..." << std::endl;
    //     init_global_XYZEW_V();
    //     std::cout << "Testing verify_initialization..." << std::endl;
    //     bool ok = verify_initialization();
    //     if (ok) {
    //         std::cout << "Initialization test PASSED." << std::endl;
    //     } else {
    //         std::cout << "Initialization test FAILED." << std::endl;
    //     }

    //     clean_global_XYZEW_V();
    //     return ok ? 0 : 1;
    // }






